#include "hip/hip_runtime.h"
/**
 **  \mainpage Mesh-based Monte Carlo (MMC) - a 3D photon simulator
 **
 **  \author Qianqian Fang <q.fang at neu.edu>
 **
 **  \section sref Reference:
 **  \li \c (\b Fang2010) Qianqian Fang, <a
 *href="http://www.opticsinfobase.org/abstract.cfm?uri=boe-1-1-165">
 **          "Mesh-based Monte Carlo Method Using Fast Ray-Tracing
 **          in Pluker Coordinates,"</a> Biomed. Opt. Express, 1(1) 165-175
 *(2010).
 **  \li \c (\b Fang2009) Qianqian Fang and David A. Boas,
 **          <a
 *href="http://www.opticsinfobase.org/abstract.cfm?uri=oe-17-22-20178">
 **          "Monte Carlo Simulation of Photon Migration in 3D Turbid Media
 *Accelerated
 **          by Graphics Processing Units,"</a> Optics Express, 17(22)
 *20178-20190 (2009).
 **
 **  \section slicense License
 **          GPL v3, see LICENSE.txt for details
 *******************************************************************************/

/***************************************************************************//**
\file    mmc_cu_host.c

\brief   CUDA host code for NVIDIA GPUs
*******************************************************************************/


#define inlinefun __device__

#include "mmc_const.h"
#include "mmc_cu_host.h"
#include "mmc_tictoc.h"
#include <stdint.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#ifdef _OPENMP
    #include <omp.h>
#endif
#include "mmc_const.h"

#include "mmc_core.cu"

/******************************************************************************
In this unit, we first launch a master thread and initialize the necessary data
structures.This include the command line options(cfg), tetrahedral mesh(mesh)
and the ray tracer precomputed data (tracer).
******************************************************************************/
#define CUDA_ASSERT(a)                                                         \
    mcx_cu_assess((a), __FILE__, __LINE__) ///< macro to report CUDA error

/**
 * @brief Utility function to calculate the GPU stream processors (cores) per SM
 *
 * Obtain GPU core number per MP, this replaces
 * ConvertSMVer2Cores() in libcudautils to avoid
 * extra dependency.
 *
 * @param[in] v1: the major version of an NVIDIA GPU
 * @param[in] v2: the minor version of an NVIDIA GPU
 */

int mcx_corecount(int v1, int v2) {
    int v = v1 * 10 + v2;

    if (v < 20) {
        return 8;
    } else if (v < 21) {
        return 32;
    } else if (v < 30) {
        return 48;
    } else if (v < 50) {
        return 192;
    } else if (v < 60 || v == 61 || v >= 89) {
        return 128;
    } else {
        return 64;
    }
}

int mcx_smxblock(int v1, int v2) {
    int v = v1 * 10 + v2;

    if (v < 30) {
        return 8;
    } else if (v < 50) {
        return 16;
    } else {
        return 32;
    }
}

/**
  assert cuda memory allocation result
 */
void mcx_cu_assess(hipError_t cuerr, const char* file, const int linenum) {
    if (cuerr != hipSuccess) {
        mcx_error(-(int)cuerr, (char*)hipGetErrorString(cuerr), file, linenum);
    }
}

/*
   master driver code to run MC simulations
*/
int mcx_list_cu_gpu(mcconfig* cfg, GPUInfo** info) {
#if __DEVICE_EMULATION__
    return 1;
#else
    int dev;
    int deviceCount, activedev = 0;

    CUDA_ASSERT(hipGetDeviceCount(&deviceCount));

    if (deviceCount == 0) {
        MMC_FPRINTF(stderr,
                    S_RED "ERROR: No CUDA-capable GPU device found\n" S_RESET);
        return 0;
    }

    *info = (GPUInfo*)calloc(deviceCount, sizeof(GPUInfo));

    if (cfg->gpuid && cfg->gpuid > (uint)deviceCount) {
        MMC_FPRINTF(stderr,
                    S_RED "ERROR: Specified GPU ID is out of range\n" S_RESET);
        return 0;
    }

    // scan from the first device
    for (dev = 0; dev < deviceCount; dev++) {
        hipDeviceProp_t dp;
        CUDA_ASSERT(hipGetDeviceProperties(&dp, dev));

        if (cfg->isgpuinfo == 3) {
            activedev++;
        } else if (cfg->deviceid[dev] == '1') {
            cfg->deviceid[dev] = '\0';
            cfg->deviceid[activedev] = dev + 1;
            activedev++;
        }

        strncpy((*info)[dev].name, dp.name, MAX_SESSION_LENGTH);
        (*info)[dev].id = dev + 1;
        (*info)[dev].devcount = deviceCount;
        (*info)[dev].major = dp.major;
        (*info)[dev].minor = dp.minor;
        (*info)[dev].globalmem = dp.totalGlobalMem;
        (*info)[dev].constmem = dp.totalConstMem;
        (*info)[dev].sharedmem = dp.sharedMemPerBlock;
        (*info)[dev].regcount = dp.regsPerBlock;
        (*info)[dev].clock = dp.clockRate;
        (*info)[dev].sm = dp.multiProcessorCount;
        (*info)[dev].core =
            dp.multiProcessorCount * mcx_corecount(dp.major, dp.minor);
        (*info)[dev].maxmpthread = dp.maxThreadsPerMultiProcessor;
        (*info)[dev].maxgate = cfg->maxgate;
        (*info)[dev].autoblock =
            (*info)[dev].maxmpthread / mcx_smxblock(dp.major, dp.minor);
        (*info)[dev].autothread = (*info)[dev].autoblock *
                                  mcx_smxblock(dp.major, dp.minor) *
                                  (*info)[dev].sm;

        if (strncmp(dp.name, "Device Emulation", 16)) {
            if (cfg->isgpuinfo) {
                MMC_FPRINTF(stdout,
                            S_BLUE "=============================   GPU Infomation  ================================\n" S_RESET);
                MMC_FPRINTF(stdout, "Device %d of %d:\t\t%s\n", (*info)[dev].id,
                            (*info)[dev].devcount, (*info)[dev].name);
                MMC_FPRINTF(stdout, "Compute Capability:\t%u.%u\n", (*info)[dev].major,
                            (*info)[dev].minor);
                MMC_FPRINTF(stdout,
                            "Global Memory:\t\t%u B\nConstant Memory:\t%u B\n"
                            "Shared Memory:\t\t%u B\nRegisters:\t\t%u\nClock "
                            "Speed:\t\t%.2f GHz\n",
                            (unsigned int)(*info)[dev].globalmem,
                            (unsigned int)(*info)[dev].constmem,
                            (unsigned int)(*info)[dev].sharedmem,
                            (unsigned int)(*info)[dev].regcount,
                            (*info)[dev].clock * 1e-6f);
#if CUDART_VERSION >= 2000
                MMC_FPRINTF(stdout, "Number of MPs:\t\t%u\nNumber of Cores:\t%u\n",
                            (*info)[dev].sm, (*info)[dev].core);
#endif
                MMC_FPRINTF(stdout, "SMX count:\t\t%u\n", (*info)[dev].sm);
            }
        }
    }

    if (cfg->isgpuinfo == 2 &&
            cfg->parentid == mpStandalone) { // list GPU info only
        exit(0);
    }

    if (activedev < MAX_DEVICE) {
        cfg->deviceid[activedev] = '\0';
    }

    return activedev;
#endif
}

void mmc_run_simulation(mcconfig* cfg, tetmesh* mesh, raytracer* tracer, GPUInfo* gpu) {
    uint i, j;
    float t, twindow0, twindow1;
    float fullload = 0.f;
    float* energy;

    uint detected = 0;
    int gpuid, threadid = 0;
    uint tic, tic0, tic1, toc = 0, fieldlen, debuglen = MCX_DEBUG_REC_LEN;
    int threadphoton, oddphotons;
    dim3 mcgrid, mcblock;

    float3* gnode;
    int4* gelem, *gfacenb;
    float4* gnormal;
    int* gtype, *gsrcelem;
    uint* gseed, *gdetected;
    volatile int* progress, *gprogress;
    float* gweight, *gdref, *gdetphoton, *genergy, *gsrcpattern, *gdebugdata;
    RandType* gphotonseed = NULL, *greplayseed = NULL;
    float*  greplayweight = NULL, *greplaytime = NULL;

    MCXReporter* greporter;
    uint meshlen = ((cfg->method == rtBLBadouelGrid) ? cfg->crop0.z : mesh->ne) * cfg->srcnum;
    cfg->crop0.w = meshlen * cfg->maxgate; // offset for the second buffer

    float* field, *dref = NULL;

    uint* Pseed = NULL;
    float* Pdet = NULL;
    RandType* Pphotonseed = NULL;

    uint detreclen = (2 + ((cfg->ismomentum) > 0)) * mesh->prop +
                     (cfg->issaveexit > 0) * 6 + 1;
    uint hostdetreclen = detreclen + 1;
    // launch mcxkernel
    size_t sharedmemsize = 0;
    double energytot = 0.0, energyesc = 0.0;

    MCXParam param = {
        cfg->srcparam1,
        cfg->srcparam2,
        cfg->crop0,
        cfg->bary0,
        make_float3(cfg->srcpos.x, cfg->srcpos.y, cfg->srcpos.z),
        make_float3(cfg->srcdir.x, cfg->srcdir.y, cfg->srcdir.z),
        make_float3(mesh->nmin.x, mesh->nmin.y, mesh->nmin.z),
        cfg->tstart,
        cfg->tend,
        (uint)cfg->isreflect,
        (uint)cfg->issavedet,
        (uint)cfg->issaveexit,
        (uint)cfg->ismomentum,
        (uint)cfg->isatomic,
        (uint)cfg->isspecular,
        1.f / cfg->tstep,
        cfg->minenergy,
        cfg->maxdetphoton,
        (uint)mesh->prop,
        (uint)cfg->detnum,
        (int)cfg->voidtime,
        (int)cfg->srctype,
        (uint)cfg->issaveref,
        (uint)cfg->maxgate,
        (uint)cfg->debuglevel,
        (int)detreclen,
        cfg->outputtype,
        mesh->elemlen,
        cfg->mcmethod,
        cfg->method,
        1.f / cfg->steps.x,
        cfg->srcdir.w,
        mesh->nn,
        mesh->ne,
        mesh->nf,
        cfg->nout,
        cfg->roulettesize,
        cfg->srcnum,
        mesh->srcelemlen,
        cfg->e0,
        cfg->isextdet,
        (uint)(meshlen / cfg->srcnum),
        (uint)(mesh->prop + 1 + cfg->isextdet) + cfg->detnum,
        (uint)(MIN((MAX_PROP - (mesh->prop + 1 + cfg->isextdet) - cfg->detnum), ((mesh->ne) << 2)) >> 2), /*max count of elem normal data in const mem*/
        cfg->issaveseed,
        cfg->seed,
        cfg->maxjumpdebug
    };

    MCXReporter reporter = {0.f, 0};

    if (cfg->issavedet) {
        sharedmemsize = sizeof(float) * detreclen;
    }

    param.reclen = sharedmemsize / sizeof(float);  //< the shared memory buffer length associated with detected photon

    sharedmemsize += sizeof(float) * (cfg->srcnum << 1);   /**< store energyesc/energytot */

    if (cfg->srctype == stPattern && cfg->srcnum > 1) {
        sharedmemsize += sizeof(float) * cfg->srcnum;
    }

    gpuid = cfg->deviceid[threadid] - 1;

    sharedmemsize *= ((int)gpu[gpuid].autoblock);

#ifdef _OPENMP
    threadid = omp_get_thread_num();
#endif

    if (threadid < MAX_DEVICE && cfg->deviceid[threadid] == '\0') {
        return;
    }

    if (gpuid < 0) {
        mcx_error(-1, "GPU ID must be non-zero", __FILE__, __LINE__);
    }

    CUDA_ASSERT(hipSetDevice(gpuid));

    #pragma omp master
    {
        if (cfg->exportfield == NULL) {
            cfg->exportfield = mesh->weight;
        }

        if (cfg->exportdetected == NULL) {
            cfg->exportdetected = (float*)malloc(hostdetreclen * cfg->maxdetphoton * sizeof(float));
        }

        if (cfg->issaveseed && cfg->exportseed == NULL) {
            cfg->exportseed = (unsigned char*)malloc(cfg->maxdetphoton * (sizeof(RandType) * RAND_BUF_LEN));
        }

        cfg->energytot = (double*)calloc(cfg->srcnum, sizeof(double));
        cfg->energyesc = (double*)calloc(cfg->srcnum, sizeof(double));
        cfg->runtime = 0;
    }
    #pragma omp barrier

    if (gpu[gpuid].maxgate == 0 && meshlen > 0) {
        int needmem = meshlen + gpu[gpuid].autothread * sizeof(float4) * 4 +
                      sizeof(float) * cfg->maxdetphoton * hostdetreclen +
                      10 * 1024 * 1024; /*keep 10M for other things*/
        gpu[gpuid].maxgate = (gpu[gpuid].globalmem - needmem) / meshlen;
        gpu[gpuid].maxgate =
            MIN(((cfg->tend - cfg->tstart) / cfg->tstep + 0.5), gpu[gpuid].maxgate);
    }

    if (!cfg->autopilot) {
        uint gates = (uint)((cfg->tend - cfg->tstart) / cfg->tstep + 0.5);
        gpu[gpuid].autothread = cfg->nthread;
        gpu[gpuid].autoblock = cfg->nblocksize;

        if (cfg->maxgate == 0) {
            cfg->maxgate = gates;
        } else if ((uint)cfg->maxgate > gates) {
            cfg->maxgate = gates;
        }

        gpu[gpuid].maxgate = cfg->maxgate;
    }

    if (gpu[gpuid].autothread % gpu[gpuid].autoblock) {
        gpu[gpuid].autothread =
            (gpu[gpuid].autothread / gpu[gpuid].autoblock) * gpu[gpuid].autoblock;
    }

    param.maxgate = gpu[gpuid].maxgate;

    uint nflen = mesh->nf * cfg->maxgate;
    #pragma omp master
    fullload = 0.f;

    for (i = 0; cfg->deviceid[i]; i++) {
        fullload += cfg->workload[i];
    }

    if (fullload < EPS) {
        for (i = 0; cfg->deviceid[i]; i++) {
            cfg->workload[i] = gpu[cfg->deviceid[i] - 1].core;
        }

    }

    #pragma omp barrier
    fullload = 0.f;

    for (i = 0; cfg->deviceid[i]; i++) {
        if (cfg->workload[i] > 0.f) {
            fullload += cfg->workload[i];
        } else {
            mcx_error(-1, "workload was unspecified for an active device", __FILE__, __LINE__);
        }
    }

    threadphoton = (int)(cfg->nphoton * cfg->workload[gpuid] /
                         (fullload * gpu[gpuid].autothread * cfg->respin));
    oddphotons =
        (int)(cfg->nphoton * cfg->workload[gpuid] / (fullload * cfg->respin) -
              threadphoton * gpu[gpuid].autothread);
    field = (float*)calloc(sizeof(float) * meshlen * 2, cfg->maxgate);
    dref = (float*)calloc(sizeof(float) * mesh->nf, cfg->maxgate);
    Pdet = (float*)calloc(cfg->maxdetphoton * sizeof(float), hostdetreclen);

    mcgrid.x = gpu[gpuid].autothread / gpu[gpuid].autoblock;
    mcblock.x = gpu[gpuid].autoblock;
    fieldlen = meshlen * cfg->maxgate;

    if (cfg->seed > 0) {
        srand(cfg->seed);
    } else {
        srand(time(0));
    }

    // create gpu pointer
    // gnode,gelem,gtype,gfacenb,gsrcelem,gnormal,gdetpos,gproperty and copy the
    // data from cpu to gpu
    CUDA_ASSERT(hipMalloc((void**)&gnode, sizeof(float3) * (mesh->nn)));
    CUDA_ASSERT(hipMemcpy(gnode, mesh->node, sizeof(float3) * (mesh->nn),
                           hipMemcpyHostToDevice));

    CUDA_ASSERT(hipMalloc((void**)&gelem, sizeof(int4) * (mesh->ne)));
    CUDA_ASSERT(hipMemcpy(gelem, mesh->elem, sizeof(int4) * (mesh->ne),
                           hipMemcpyHostToDevice));

    CUDA_ASSERT(hipMalloc((void**)&gtype, sizeof(int) * (mesh->ne)));
    CUDA_ASSERT(hipMemcpy(gtype, mesh->type, sizeof(int) * (mesh->ne),
                           hipMemcpyHostToDevice));

    CUDA_ASSERT(hipMalloc((void**)&gfacenb, sizeof(int4) * (mesh->ne)));
    CUDA_ASSERT(hipMemcpy(gfacenb, mesh->facenb, sizeof(int4) * (mesh->ne),
                           hipMemcpyHostToDevice));

    if (mesh->srcelemlen > 0) {
        CUDA_ASSERT(hipMalloc((void**)&gsrcelem, sizeof(int) * (mesh->srcelemlen)));
        CUDA_ASSERT(hipMemcpy(gsrcelem, mesh->srcelem,
                               sizeof(int) * (mesh->srcelemlen),
                               hipMemcpyHostToDevice));
    } else {
        gsrcelem = NULL;
    }

    CUDA_ASSERT(hipMalloc((void**)&gnormal, sizeof(float4) * (mesh->ne) * 4));
    CUDA_ASSERT(hipMemcpy(gnormal, tracer->n, sizeof(float4) * (mesh->ne) * 4,
                           hipMemcpyHostToDevice));

    // gparam
    CUDA_ASSERT(hipMemcpyToSymbol(HIP_SYMBOL(gcfg), &param, sizeof(MCXParam), 0, hipMemcpyHostToDevice));
    CUDA_ASSERT(hipMemcpyToSymbol(HIP_SYMBOL(gmed), mesh->med,
                                   (mesh->prop + 1 + cfg->isextdet) * sizeof(Medium), 0,
                                   hipMemcpyHostToDevice));

    if (cfg->detpos && cfg->detnum) {
        if ((mesh->prop + 1 + cfg->isextdet) + cfg->detnum >= MAX_PROP) {
            mcx_error(-5, "Total tissue type and detector count must be less than 2000", __FILE__, __LINE__);
        }

        CUDA_ASSERT(hipMemcpyToSymbol(HIP_SYMBOL(gmed), cfg->detpos,
                                       sizeof(float4)*cfg->detnum, (mesh->prop + 1 + cfg->isextdet) * sizeof(Medium),
                                       hipMemcpyHostToDevice));
    }

    CUDA_ASSERT(hipMemcpyToSymbol(HIP_SYMBOL(gmed), tracer->n,
                                   (param.normbuf << 2) * (sizeof(float4)), sizeof(float4)*param.maxpropdet,
                                   hipMemcpyHostToDevice));

    // gprogress
    CUDA_ASSERT(
        hipHostAlloc((void**)&progress, sizeof(int), hipHostMallocMapped));
    CUDA_ASSERT(hipHostGetDevicePointer((int**)&gprogress, (int*)progress, 0));
    *progress = 0;

    Pseed = (uint*)malloc(sizeof(uint) * gpu[gpuid].autothread * RAND_SEED_WORD_LEN);
    energy = (float*)calloc(sizeof(float) * cfg->srcnum, gpu[gpuid].autothread << 1);

    for (j = 0; j < gpu[gpuid].autothread * RAND_SEED_WORD_LEN; j++) {
        Pseed[j] = rand();
    }

    CUDA_ASSERT(hipMalloc((void**)&gseed, sizeof(uint) * gpu[gpuid].autothread*
                           RAND_SEED_WORD_LEN));
    CUDA_ASSERT(hipMemcpy(
                    gseed, Pseed, sizeof(uint) * gpu[gpuid].autothread * RAND_SEED_WORD_LEN,
                    hipMemcpyHostToDevice));

    CUDA_ASSERT(hipMalloc((void**)&gweight, sizeof(float) * fieldlen * 2));
    CUDA_ASSERT(hipMemcpy(gweight, field, sizeof(float) * fieldlen * 2,
                           hipMemcpyHostToDevice));

    CUDA_ASSERT(hipMalloc((void**)&gdref, sizeof(float) * nflen));
    CUDA_ASSERT(
        hipMemcpy(gdref, dref, sizeof(float) * nflen, hipMemcpyHostToDevice));

    CUDA_ASSERT(hipMalloc((void**)&gdetphoton,
                           sizeof(float) * cfg->maxdetphoton * hostdetreclen));
    CUDA_ASSERT(hipMemcpy(gdetphoton, Pdet,
                           sizeof(float) * cfg->maxdetphoton * hostdetreclen,
                           hipMemcpyHostToDevice));

    CUDA_ASSERT(hipMalloc((void**)&genergy,
                           sizeof(float) * (gpu[gpuid].autothread << 1) * cfg->srcnum));
    CUDA_ASSERT(hipMemcpy(genergy, energy,
                           sizeof(float) * (gpu[gpuid].autothread << 1) * cfg->srcnum,
                           hipMemcpyHostToDevice));

    CUDA_ASSERT(hipMalloc((void**)&gdetected, sizeof(uint)));
    CUDA_ASSERT(hipMemcpy(gdetected, &detected, sizeof(uint), hipMemcpyHostToDevice));

    CUDA_ASSERT(hipMalloc((void**)&greporter, sizeof(MCXReporter)));
    CUDA_ASSERT(hipMemcpy(greporter, &reporter, sizeof(MCXReporter),
                           hipMemcpyHostToDevice));

    if (cfg->srctype == MCX_SRC_PATTERN) {
        CUDA_ASSERT(hipMalloc((void**)&gsrcpattern,
                               sizeof(float) * (int)(cfg->srcparam1.w * cfg->srcparam2.w * cfg->srcnum)));
        CUDA_ASSERT(hipMemcpy(gsrcpattern, cfg->srcpattern,
                               sizeof(float) * (int)(cfg->srcparam1.w * cfg->srcparam2.w * cfg->srcnum),
                               hipMemcpyHostToDevice));
    } else if (cfg->srctype == MCX_SRC_PATTERN3D) {
        CUDA_ASSERT(hipMalloc((void**)&gsrcpattern,
                               sizeof(float) * (int)(cfg->srcparam1.x * cfg->srcparam1.y*
                                       cfg->srcparam1.z * cfg->srcnum)));
        CUDA_ASSERT(hipMemcpy(gsrcpattern, cfg->srcpattern,
                               sizeof(float) * (int)(cfg->srcparam1.x * cfg->srcparam1.y*
                                       cfg->srcparam1.z * cfg->srcnum),
                               hipMemcpyHostToDevice));
    } else {
        gsrcpattern = NULL;
    }

    if (cfg->issaveseed) {
        Pphotonseed = (RandType*)calloc(cfg->maxdetphoton, (sizeof(RandType) * RAND_BUF_LEN));
        CUDA_ASSERT(hipMalloc((void**)&gphotonseed, cfg->maxdetphoton * (sizeof(RandType)*RAND_BUF_LEN)));
    }

    if (cfg->debuglevel & dlTraj) {
        CUDA_ASSERT(hipMalloc((void**) &gdebugdata, sizeof(float) * (debuglen * cfg->maxjumpdebug)));
    }

    if (cfg->seed == SEED_FROM_FILE) {
        CUDA_ASSERT(hipMalloc((void**)&greplayweight, sizeof(float)*cfg->nphoton));
        CUDA_ASSERT(hipMemcpy(greplayweight, cfg->replayweight, sizeof(float)*cfg->nphoton, hipMemcpyHostToDevice));

        CUDA_ASSERT(hipMalloc((void**)&greplaytime, sizeof(float)*cfg->nphoton));
        CUDA_ASSERT(hipMemcpy(greplaytime, cfg->replaytime, sizeof(float)*cfg->nphoton, hipMemcpyHostToDevice));

        CUDA_ASSERT(hipMalloc((void**)&greplayseed, (sizeof(RandType)*RAND_BUF_LEN)*cfg->nphoton));
        CUDA_ASSERT(hipMemcpy(greplayseed, cfg->photonseed, (sizeof(RandType)*RAND_BUF_LEN)*cfg->nphoton, hipMemcpyHostToDevice));
    }

    free(Pseed);
    free(energy);
    tic = StartTimer();

    #pragma omp master
    {
        mcx_printheader(cfg);

#ifdef MCX_TARGET_NAME
        MMC_FPRINTF(
            cfg->flog, "- code name: [%s] compiled by nvcc [%d.%d] with CUDA [%d]\n",
            "MMC-Trinity", __CUDACC_VER_MAJOR__, __CUDACC_VER_MINOR__, CUDART_VERSION);
#else
        MMC_FPRINTF(
            cfg->flog, "- code name: [MMC-Trinity] compiled by nvcc [%d.%d] with CUDA [%d]\n",
            __CUDACC_VER_MAJOR__, __CUDACC_VER_MINOR__, CUDART_VERSION);
#endif
        MMC_FPRINTF(cfg->flog, "- compiled with: [RNG] %s [Seed Length] %d\n",
                    MCX_RNG_NAME, RAND_SEED_WORD_LEN);
        mcx_fflush(cfg->flog);
    }
    #pragma omp barrier

    MMC_FPRINTF(cfg->flog,
                "- [device %d(%d): %s] threadph=%d oddphotons=%d np=%.1f "
                "nthread=%d nblock=%d repetition=%d\n",
                gpuid + 1, gpu[gpuid].id, gpu[gpuid].name, threadphoton,
                oddphotons, cfg->nphoton * cfg->workload[gpuid] / fullload,
                (int)gpu[gpuid].autothread, (int)gpu[gpuid].autoblock,
                cfg->respin);

    // simulate for all time-gates in maxgate groups per run

    tic0 = GetTimeMillis();

    for (t = cfg->tstart; t < cfg->tend; t += cfg->tstep * cfg->maxgate) {
        twindow0 = t;
        twindow1 = t + cfg->tstep * cfg->maxgate;

        MMC_FPRINTF(cfg->flog,
                    "lauching mcx_main_loop for time window [%.1fns %.1fns] ...\n",
                    twindow0 * 1e9, twindow1 * 1e9);

        MMC_FPRINTF(cfg->flog, "requesting %ld bytes of shared memory\n", sharedmemsize);

        mcx_fflush(cfg->flog);

        // total number of repetition for the simulations, results will be
        // accumulated to field
        for (int iter = 0; iter < cfg->respin; iter++) {
            MMC_FPRINTF(cfg->flog, "simulation run#%2d ... \n", iter + 1);
            mcx_fflush(cfg->flog);
            mcx_fflush(cfg->flog);
            param.tstart = twindow0;
            param.tend = twindow1;


            mmc_main_loop <<< mcgrid, mcblock, sharedmemsize>>>(
                threadphoton, oddphotons, gnode, (int*)gelem, gweight, gdref,
                gtype, (int*)gfacenb, gsrcelem, gnormal,
                gdetphoton, gdetected, gseed, (int*)gprogress, genergy, greporter,
                gsrcpattern, greplayweight, greplaytime, greplayseed, gphotonseed, gdebugdata);

            #pragma omp master
            {
                if ((cfg->debuglevel & MCX_DEBUG_PROGRESS)) {
                    int p0 = 0, ndone = -1;

                    mcx_progressbar(-0.f);

                    do {
                        ndone = *progress;

                        if (ndone > p0) {
                            mcx_progressbar((float)ndone / gpu[0].autothread);
                            p0 = ndone;
                        }

                        sleep_ms(100);
                    } while (p0 < (int)gpu[0].autothread);

                    mcx_progressbar(1.f);
                    MMC_FPRINTF(cfg->flog, "\n");
                }
            }
            CUDA_ASSERT(hipDeviceSynchronize());
            tic1 = GetTimeMillis();
            toc += tic1 - tic0;
            MMC_FPRINTF(cfg->flog,
                        "kernel complete:  \t%d ms\nretrieving flux ... \t",
                        tic1 - tic);
            mcx_fflush(cfg->flog);
            #pragma omp critical

            if (cfg->runtime < tic1 - tic) {
                cfg->runtime = tic1 - tic;
            }

            MCXReporter rep;
            CUDA_ASSERT(hipMemcpy(&rep, greporter, sizeof(MCXReporter),
                                   hipMemcpyDeviceToHost));
            reporter.raytet += rep.raytet;
            reporter.jumpdebug += rep.jumpdebug;

            energy = (float*)calloc(sizeof(float) * cfg->srcnum, gpu[gpuid].autothread << 1);

            CUDA_ASSERT(hipMemcpy(energy, genergy,
                                   sizeof(float) * (gpu[gpuid].autothread << 1) * cfg->srcnum,
                                   hipMemcpyDeviceToHost));
            #pragma omp critical
            {

                for (i = 0; i < gpu[gpuid].autothread; i++) {
                    for (j = 0; j < (uint) cfg->srcnum; j++) {
                        cfg->energyesc[j] += energy[(i << 1) * cfg->srcnum + j];
                        cfg->energytot[j] += energy[((i << 1) + 1) * cfg->srcnum + j];
                        energyesc += energy[(i << 1) * cfg->srcnum + j];
                        energytot += energy[((i << 1) + 1) * cfg->srcnum + j];
                    }
                }
            }

            free(energy);

            /**
             * If '-D M' is specified, we retrieve photon trajectory data and store those to \c cfg.exportdebugdata and \c cfg.debugdatalen
             */
            if (cfg->debuglevel & dlTraj) {
                uint debugrec = reporter.jumpdebug;

                #pragma omp critical
                {
                    if (debugrec > 0) {
                        if (debugrec > cfg->maxjumpdebug) {
                            MMC_FPRINTF(cfg->flog, S_RED "WARNING: the saved trajectory positions (%d) \
are more than what your have specified (%d), please use the --maxjumpdebug option to specify a greater number\n" S_RESET
                                        , debugrec, cfg->maxjumpdebug);
                        } else {
                            MMC_FPRINTF(cfg->flog, "saved %u trajectory positions, total: %d\t", debugrec, cfg->debugdatalen + debugrec);
                        }

                        debugrec = min(debugrec, cfg->maxjumpdebug);
                        cfg->exportdebugdata = (float*)realloc(cfg->exportdebugdata, (cfg->debugdatalen + debugrec) * debuglen * sizeof(float));
                        CUDA_ASSERT(hipMemcpy(cfg->exportdebugdata + cfg->debugdatalen, gdebugdata, sizeof(float)*debuglen * debugrec, hipMemcpyDeviceToHost));
                        cfg->debugdatalen += debugrec;
                    }
                }
            }

            if (cfg->issavedet) {
                CUDA_ASSERT(hipMemcpy(&detected, gdetected, sizeof(uint), hipMemcpyDeviceToHost));

                CUDA_ASSERT(hipMemcpy(Pdet, gdetphoton, sizeof(float) * cfg->maxdetphoton * hostdetreclen,
                                       hipMemcpyDeviceToHost));

                if (cfg->issaveseed) {
                    CUDA_ASSERT(hipMemcpy(Pphotonseed, gphotonseed, cfg->maxdetphoton * (sizeof(RandType)*RAND_BUF_LEN),
                                           hipMemcpyDeviceToHost));
                }

                if (detected > cfg->maxdetphoton) {
                    MMC_FPRINTF(cfg->flog, "WARNING: the detected photon (%d) \
              is more than what your have specified (%d), please use the -H option to specify a greater number\t",
                                detected, cfg->maxdetphoton);
                } else {
                    MMC_FPRINTF(cfg->flog, "detected %d photons, total: %d\t", detected,
                                cfg->detectedcount + detected);
                }

                #pragma omp atomic
                cfg->his.detected += detected;
                detected = MIN(detected, cfg->maxdetphoton);

                if (cfg->exportdetected) {
                    #pragma omp critical
                    {
                        cfg->exportdetected = (float*)realloc(
                            cfg->exportdetected, (cfg->detectedcount + detected) *
                            hostdetreclen * sizeof(float));
                        memcpy(cfg->exportdetected + cfg->detectedcount * (hostdetreclen),
                               Pdet, detected * (hostdetreclen) * sizeof(float));

                        if (cfg->issaveseed) {
                            cfg->exportseed = (unsigned char*)realloc(cfg->exportseed, (cfg->detectedcount + detected) * (sizeof(RandType) * RAND_BUF_LEN));
                            memcpy(cfg->exportseed + cfg->detectedcount * sizeof(RandType)*RAND_BUF_LEN, Pphotonseed, detected * (sizeof(RandType)*RAND_BUF_LEN));
                        }

                        cfg->detectedcount += detected;
                    }
                }
            }

            if (cfg->issaveref) {
                float* rawdref = (float*)calloc(sizeof(float), nflen);

                CUDA_ASSERT(hipMemcpy(rawdref, gdref, sizeof(float) * nflen,
                                       hipMemcpyDeviceToHost));

                for (i = 0; i < nflen; i++) { // accumulate field, can be done in the GPU
                    dref[i] += rawdref[i];    //+rawfield[i+fieldlen];
                }

                free(rawdref);
            }

            // handling the 2pt distributions
            if (cfg->issave2pt) {
                float* rawfield = (float*)malloc(sizeof(float) * fieldlen * 2);

                CUDA_ASSERT(hipMemcpy(rawfield, gweight, sizeof(float) * fieldlen * 2,
                                       hipMemcpyDeviceToHost));
                MMC_FPRINTF(cfg->flog, "transfer complete:        %d ms\n",
                            GetTimeMillis() - tic);
                mcx_fflush(cfg->flog);

                for (i = 0; i < fieldlen; i++) { // accumulate field, can be done in the GPU
                    field[i] += rawfield[i] + rawfield[i + fieldlen];    //+rawfield[i+fieldlen];
                }

                free(rawfield);
            }

            if (cfg->respin > 1 && RAND_SEED_WORD_LEN > 1) {
                Pseed = (uint*)malloc(sizeof(uint) * gpu[gpuid].autothread *
                                      RAND_SEED_WORD_LEN);

                for (i = 0; i < gpu[gpuid].autothread * RAND_SEED_WORD_LEN; i++) {
                    Pseed[i] = rand();
                }

                CUDA_ASSERT(hipMemcpy(gseed, Pseed,
                                       sizeof(uint) * gpu[gpuid].autothread*
                                       RAND_SEED_WORD_LEN,
                                       hipMemcpyHostToDevice));
                free(Pseed);
            }

            // loop over work devices
        } // iteration
    }   // time gates

    #pragma omp master
    {
        int i, j, srcid;

        if (cfg->exportfield) {
            if (cfg->basisorder == 0 || cfg->method == rtBLBadouelGrid) {
                for (uint i = 0; i < fieldlen; i++)
                    #pragma omp atomic
                    cfg->exportfield[i] += field[i];
            } else {
                for (i = 0; i < cfg->maxgate; i++) {
                    for (j = 0; j < mesh->ne; j++) {
                        for (srcid = 0; srcid < cfg->srcnum; srcid++) {
                            float ww = field[(i * mesh->ne + j) * cfg->srcnum + srcid] * 0.25f;
                            int k;

                            for (k = 0; k < mesh->elemlen; k++) {
                                cfg->exportfield[(i * mesh->nn + mesh->elem[j * mesh->elemlen + k] - 1) * cfg->srcnum + srcid] += ww;
                            }
                        }
                    }
                }
            }
        }

        if (cfg->issaveref && mesh->dref) {
            for (uint i = 0; i < nflen; i++) {
                mesh->dref[i] += dref[i];
            }
        }

        if (cfg->isnormalized) {
            double cur_normalizer, sum_normalizer = 0.0, energyabs = 0.0;

            for (j = 0; j < cfg->srcnum; j++) {
                energyabs =  cfg->energytot[j] - cfg->energyesc[j];
                cur_normalizer = mesh_normalize(mesh, cfg, energyabs, cfg->energytot[j], j);
                sum_normalizer += cur_normalizer;
                MMCDEBUG(cfg, dlTime, (cfg->flog, "source %d\ttotal simulated energy: %f\tabsorbed: " S_BOLD "" S_BLUE "%5.5f%%" S_RESET "\tnormalizor=%g\n",
                                       j + 1, cfg->energytot[j], 100.f * energyabs / cfg->energytot[j], cur_normalizer));
            }

            cfg->his.normalizer = sum_normalizer / cfg->srcnum; // average normalizer value for all simulated sources
        }

#ifndef MCX_CONTAINER

        if (cfg->issave2pt && cfg->parentid == mpStandalone) {
            MMC_FPRINTF(cfg->flog, "saving data to file ...\t");
            mesh_saveweight(mesh, cfg, 0);
            MMC_FPRINTF(cfg->flog, "saving data complete : %d ms\n\n",
                        GetTimeMillis() - tic);
            mcx_fflush(cfg->flog);
        }

        if (cfg->issavedet && cfg->parentid == mpStandalone &&
                cfg->exportdetected) {
            cfg->his.totalphoton = cfg->nphoton;
            cfg->his.unitinmm = cfg->unitinmm;
            cfg->his.savedphoton = cfg->detectedcount;
            cfg->his.detected = cfg->detectedcount;
            cfg->his.colcount = (2 + (cfg->ismomentum > 0)) * cfg->his.maxmedia + (cfg->issaveexit > 0) * 6 + 2; /*column count=maxmedia+3*/
            mesh_savedetphoton(cfg->exportdetected, (void*)(cfg->exportseed), cfg->detectedcount,
                               (sizeof(uint64_t) * RAND_BUF_LEN), cfg);
        }

        /**
         * If not running as a mex file, we need to save photon trajectory data, if enabled, as
         * a file, either as a .mct file, or a .jdat/.jbat file
         */

        if ((cfg->debuglevel & dlTraj) && cfg->parentid == mpStandalone && cfg->exportdebugdata) {
            cfg->his.colcount = MCX_DEBUG_REC_LEN;
            cfg->his.savedphoton = cfg->debugdatalen;
            cfg->his.totalphoton = cfg->nphoton;
            cfg->his.detected = 0;
            mesh_savedetphoton(cfg->exportdebugdata, NULL, cfg->debugdatalen, 0, cfg);
        }

        if (cfg->issaveref) {
            MMC_FPRINTF(cfg->flog, "saving surface diffuse reflectance ...");
            mesh_saveweight(mesh, cfg, 1);
        }

#endif

        // total energy here equals total simulated photons+unfinished photons for
        // all threads
        MMC_FPRINTF(cfg->flog,
                    "simulated %ld photons (%ld) with devices (ray-tet "
                    "%.0f)\nMCX simulation speed: %.2f photon/ms\n",
                    cfg->nphoton, cfg->nphoton, reporter.raytet,
                    (double)cfg->nphoton / toc);
        MMC_FPRINTF(cfg->flog,
                    "total simulated energy: %.2f\tabsorbed: %5.5f%%\n(loss due to "
                    "initial specular reflection is excluded in the total)\n",
                    energytot,
                    (energytot - energyesc) / energytot * 100.f);
        mcx_fflush(cfg->flog);
    }
    #pragma omp barrier
    CUDA_ASSERT(hipFree(gnode));
    CUDA_ASSERT(hipFree(gelem));
    CUDA_ASSERT(hipFree(gtype));
    CUDA_ASSERT(hipFree(gfacenb));
    CUDA_ASSERT(hipFree(gsrcelem));
    CUDA_ASSERT(hipFree(gnormal));
    CUDA_ASSERT(hipFree(gseed));
    CUDA_ASSERT(hipFree(gdetphoton));
    CUDA_ASSERT(hipFree(gweight));
    CUDA_ASSERT(hipFree(gdref));
    CUDA_ASSERT(hipFree(genergy));
    CUDA_ASSERT(hipFree(gdetected));

    if (gsrcpattern) {
        CUDA_ASSERT(hipFree(gsrcpattern));
    }

    if (greplayweight) {
        CUDA_ASSERT(hipFree(greplayweight));
    }

    if (greplayseed) {
        CUDA_ASSERT(hipFree(greplayseed));
    }

    if (greplaytime) {
        CUDA_ASSERT(hipFree(greplaytime));
    }

    if (gphotonseed) {
        CUDA_ASSERT(hipFree(gphotonseed));
    }

    if (cfg->debuglevel & dlTraj) {
        CUDA_ASSERT(hipFree(gdebugdata));
    }

    CUDA_ASSERT(hipFree(greporter));

    #pragma omp master
    {
        if (gpu) {
            free(gpu);
        }

        free(cfg->energytot);
        free(cfg->energyesc);
        cfg->energytot = NULL;
        cfg->energyesc = NULL;
    }

    free(field);

    if (Pdet) {
        free(Pdet);
    }

    if (Pphotonseed) {
        free(Pphotonseed);
    }

    free(dref);
}

void mmc_run_cu(mcconfig* cfg, tetmesh* mesh, raytracer* tracer) {
    GPUInfo* gpuinfo = NULL;      /** gpuinfo: structure to store GPU information */
    unsigned int activedev = 0;   /** activedev: count of total active GPUs to be used */

    if (!(activedev = mcx_list_cu_gpu(cfg, &gpuinfo))) {
        mcx_error(-1, "No GPU device found\n", __FILE__, __LINE__);
    }

#ifdef _OPENMP
    /**
        Now we are ready to launch one thread for each involked GPU to run the simulation
     */
    omp_set_num_threads(activedev);
    #pragma omp parallel
    {
#endif

        /**
            This line runs the main MCX simulation for each GPU inside each thread
         */
        mmc_run_simulation(cfg, mesh, tracer, gpuinfo);

#ifdef _OPENMP
    }
#endif
}
